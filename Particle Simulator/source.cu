#include "hip/hip_runtime.h"
#include "imgui.h"
#include"imgui_impl_glfw.h"
#include "imgui_impl_opengl3.h"

#include "glad.h"
#include <GLFW/glfw3.h>
#include <iostream>

#include <fstream>
#include <string>

#include "glm/glm.hpp"
#include "glm/gtc/matrix_transform.hpp"
#include "glm/gtc/type_ptr.hpp"


#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

#include "kernel.cuh"


const int WIDTH = 2000;
const int HEIGHT = 1200;
const float GRIDL = 800;
const int PCOUNT = 1000;

const int BLOCKSIZE = 128;
const int NUMBLOCKS = (PCOUNT + BLOCKSIZE - 1) / BLOCKSIZE;


void initParts(Particle* parts, const int PCOUNT);
void randPos(Particle* parts, const int PCOUN, const int Limit);

int main()
{
	glfwInit();
	glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 4);
	glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 6);
	glfwWindowHint(GLFW_OPENGL_PROFILE, GLFW_OPENGL_CORE_PROFILE);

	//create window
	GLFWwindow* window = glfwCreateWindow(WIDTH, HEIGHT, "N-Body Simulation", NULL, NULL);
	glfwMakeContextCurrent(window);
	if (!gladLoadGLLoader((GLADloadproc)glfwGetProcAddress)) {
		std::cout << "Failed to initialize OpenGL context" << std::endl;
		return -1;
	}

	glViewport(0, 0, WIDTH, HEIGHT);





	//glClearColor(0.07f, 0.13f, 0.17f, 1.0f);
	glClearColor(0.0f, 0.0f, 0.0f, 1.0f);

	std::string fragShaderString;
	std::string vertShaderString;

	//load shader files
	std::ifstream file("shaders/Fragment.frag");
	if (file.fail())
		std::cout << "It failed\n" << strerror(errno) << std::endl;
	if (file)
	{
		std::string contents((std::istreambuf_iterator<char>(file)),
			std::istreambuf_iterator<char>());

		fragShaderString = contents;
	}
	file.close();
	file.open("shaders/Vertex.vert");
	if (file.fail())
		std::cout << "It failed\n" << strerror(errno) << std::endl;
	if (file)
	{
		std::string contents((std::istreambuf_iterator<char>(file)),
			std::istreambuf_iterator<char>());

		vertShaderString = contents;
	}
	file.close();
	const char* vShader = vertShaderString.c_str();
	const char* fShader = fragShaderString.c_str();

	//std::cout << "Vert shader Below\n" << std::endl;
	//std::cout << vShader << std::endl;
	//std::cout << "Frag shader Below\n" << std::endl;
	//std::cout << fShader << std::endl;

	// a triangle
	float vertices[PCOUNT * 3];




	//bind shaders
	GLuint vertShader = glCreateShader(GL_VERTEX_SHADER);
	glShaderSource(vertShader, 1, &vShader, NULL);
	glCompileShader(vertShader);
	GLuint fragShader = glCreateShader(GL_FRAGMENT_SHADER);
	glShaderSource(fragShader, 1, &fShader, NULL);
	glCompileShader(fragShader);

	//create shaders to shaderProgram
	GLuint shaderProgram = glCreateProgram();
	glAttachShader(shaderProgram, vertShader);
	glAttachShader(shaderProgram, fragShader);
	glLinkProgram(shaderProgram);
	glUseProgram(shaderProgram);

	//these are not needed anymore
	glDeleteShader(vertShader);
	glDeleteShader(fragShader);

	GLuint VAO, VBO;
	glGenVertexArrays(1, &VAO);
	glGenBuffers(1, &VBO);
	//bind the vertex array to use
	glBindVertexArray(VAO);
	//bind the buffer object to use
	glBindBuffer(GL_ARRAY_BUFFER, VBO);
	glBufferData(GL_ARRAY_BUFFER, sizeof(vertices), vertices, GL_DYNAMIC_DRAW);

	//select vertex attrib to modify
	glVertexAttribPointer(0, 3, GL_FLOAT, GL_FALSE, 3 * sizeof(float), (void*)0);
	glEnableVertexAttribArray(0);

	glBindBuffer(GL_ARRAY_BUFFER, 0);
	glBindVertexArray(0);

	float rotation = 0.0f;
	double prevTime = glfwGetTime();

	glPointSize(6.0f);
	glEnable(GL_POINT_SMOOTH);
	glEnable(GL_BLEND);
	//glBlendFunc(GL_DST_ALPHA, GL_SRC_ALPHA);
	glBlendFunc(GL_SRC_ALPHA, GL_DST_ALPHA);



	//glEnable(GL_DEPTH_TEST);

	//register the buffer object to cuda memory space
	//cudaGLRegisterBufferObject(VBO);
	hipGraphicsResource* resource;
	//hipGraphicsGLRegisterBuffer(&resource, VBO, cudaGLMapFlagsNone);
	//hipGraphicsMapResources(1, &resource, 0);
	void** vertPointer;
	float3* vertsPtr;
	size_t mappedSize;
	// Map the buffer to CUDA
	//hipGraphicsResourceGetMappedPointer((void**)&vertsPtr, &mappedSize, resource);
	//cudaGLMapBufferObject(&vertPointer, VBO);
	// Run a kernel to create/manipulate the data
	//testKernal << <1, 1 >> > (vertsPtr);

	// Unmap the buffer // must be unmapped for opengl to use
	//hipDeviceSynchronize();
	//hipGraphicsUnmapResources(1, &resource,0);

	Particle* parts;
	Particle* temp;
	temp = new Particle[PCOUNT];




	hipMalloc(&parts, PCOUNT * sizeof(Particle));
	initParts(temp, PCOUNT);
	randPos(temp, PCOUNT, 80);



	hipMemcpy(parts, temp, PCOUNT * sizeof(Particle), hipMemcpyHostToDevice);

	IMGUI_CHECKVERSION();
	ImGui::CreateContext();
	ImGuiIO& io = ImGui::GetIO(); (void)io;
	ImGui::StyleColorsDark();
	ImGui_ImplGlfw_InitForOpenGL(window, true);
	ImGui_ImplOpenGL3_Init("#version 330");









	bool pause = false;
	float delta = 0.005;
	float rotateAmount = 0.2f;
	float zoom = -100.0f;

	//glfwSwapBuffers(window);
	while (!glfwWindowShouldClose(window))
	{
		glClear(GL_COLOR_BUFFER_BIT);

		ImGui_ImplOpenGL3_NewFrame();
		ImGui_ImplGlfw_NewFrame();
		ImGui::NewFrame();

		ImGui::Begin("Nope");
		ImGui::Text("Please?");
		ImGui::Checkbox("Pause", &pause);
		ImGui::SliderFloat("Delta", &delta, 0.0001f, 0.1f);
		ImGui::SliderFloat("Rotation speed", &rotateAmount, 0.0f, 1.0f);
		ImGui::SliderFloat("Zoom?", &zoom, -100.0f, 100.0f);
		ImGui::End();


		double crntTime = glfwGetTime();

		rotation += rotateAmount;
		prevTime = crntTime;

		glm::mat4 proj = glm::mat4(1.0f);
		glm::mat4 model = glm::mat4(1.0f);
		glm::mat4 view = glm::mat4(1.0f);

		proj = glm::perspective(glm::radians(60.0f), (float)WIDTH/(float)HEIGHT, 0.01f, 1000.0f);
		model = glm::rotate(model, glm::radians(rotation), glm::vec3(0.0f, 1.0f, 0.0f));
		view = glm::translate(view, glm::vec3(0.0f, 0.0f, zoom));


		int modelLoc = glGetUniformLocation(shaderProgram, "model");
		glUniformMatrix4fv(modelLoc, 1, GL_FALSE, glm::value_ptr(model));
		int viewLoc = glGetUniformLocation(shaderProgram, "view");
		glUniformMatrix4fv(viewLoc, 1, GL_FALSE, glm::value_ptr(view));
		int projLoc = glGetUniformLocation(shaderProgram, "proj");
		glUniformMatrix4fv(projLoc, 1, GL_FALSE, glm::value_ptr(proj));


		//glBindBuffer(GL_ARRAY_BUFFER, 0);
		//glBindVertexArray(0);
		hipGraphicsGLRegisterBuffer(&resource, VBO, cudaGLMapFlagsNone);
		hipGraphicsMapResources(1, &resource, 0);


		hipGraphicsResourceGetMappedPointer((void**)&vertsPtr, &mappedSize, resource);
		//cudaGLMapBufferObject(&vertPointer, VBO);
		// Run a kernel to create/manipulate the data
		//testKernal << <1, 1 >> > (vertsPtr);
		if (pause == false)
		{
			naiveNBody << <NUMBLOCKS, BLOCKSIZE >> > (parts, vertsPtr, PCOUNT, delta);
			hipDeviceSynchronize();
			updateVertexBuffer << < NUMBLOCKS, BLOCKSIZE >> > (parts, vertsPtr, PCOUNT);
		}




		// Unmap the buffer // must be unmapped for opengl to use
		//hipDeviceSynchronize();
		hipGraphicsUnmapResources(1, &resource, 0);





		ImGui::Render();
		ImGui_ImplOpenGL3_RenderDrawData(ImGui::GetDrawData());



		glBindVertexArray(VAO);

		glDrawArrays(GL_POINTS, 0, PCOUNT);
		glfwSwapBuffers(window);





		glfwPollEvents();



	}


	ImGui_ImplGlfw_Shutdown();
	ImGui_ImplOpenGL3_Shutdown();
	ImGui::DestroyContext();


	//delete objects before program close
	glDeleteVertexArrays(1, &VAO);
	glDeleteBuffers(1, &VBO);
	glDeleteProgram(shaderProgram);

	glfwDestroyWindow(window);
	glfwTerminate();
	return 0;
}
//sets all values to 0
void initParts(Particle* partsTmp, const int PCOUNT) {
	for (size_t i = 0; i < PCOUNT; i++)
	{
		partsTmp[i].velocity.x = 0;
		partsTmp[i].velocity.y = 0;
		partsTmp[i].velocity.z = 0;
		partsTmp[i].position.x = 0;
		partsTmp[i].position.y = 0;
		partsTmp[i].position.z = 0;
	}


}

void randPos(Particle* parts, const int PCOUNT, const int Limit)
{
	for (size_t i = 0; i < PCOUNT; i++)
	{
		float temp;
		temp = rand()%1000;
		temp = temp * 0.0001;



		parts[i].position.x = (rand() % Limit - Limit / 2)+temp;
		parts[i].position.y = (rand() % Limit - Limit / 2)+temp;
		parts[i].position.z = (rand() % Limit - Limit / 2)+temp;
		//std::cout << parts[i].position.x << std::endl;
	}



}
